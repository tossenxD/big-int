#include "hip/hip_runtime.h"
#include "helpers.h"
#include "kernels.cu.h"

using namespace std;

#define GPU_RUNS_ADD    300
#define WITH_VALIDATION 1
#define DEBUG           0

/****************************/
/*** Big-Integer Addition ***/
/****************************/

// wrapper that invokes and times the big int addition GPU kernels;
// m is the size in Base::uint_t units and v is the kernel version.
template<class Base, uint32_t m, uint32_t v>
void gpuAdd (uint32_t num_instances, typename Base::uint_t* h_as,
             typename Base::uint_t* h_bs, typename Base::uint_t* h_rs) {

    assert((Base::bits >= 32) && (Base::bits % 32 == 0));

    using uint_t = typename Base::uint_t;
    uint_t* d_as;
    uint_t* d_bs;
    uint_t* d_rs;
    size_t mem_size_nums = num_instances * m * sizeof(uint_t);

    // 1. allocate device memory
    hipMalloc((void**) &d_as, mem_size_nums);
    hipMalloc((void**) &d_bs, mem_size_nums);
    hipMalloc((void**) &d_rs, mem_size_nums);
 
    // 2. copy host memory to device
    hipMemcpy(d_as, h_as, mem_size_nums, hipMemcpyHostToDevice);
    hipMemcpy(d_bs, h_bs, mem_size_nums, hipMemcpyHostToDevice);

    // 3. kernel dimensions
    const uint32_t q = (v > 1) ? 4 : 1;
    assert(m%q == 0 && m >= q && m/q <= 1024);
    const uint32_t ipb = (v > 2) ? (128 + m/q - 1) / (m/q) : 1; // ceil(128/(m/q))
    dim3 block(ipb*(m/q), 1, 1);
    dim3 grid (num_instances/ipb, 1, 1);
    #if DEBUG
    printf("\n[debug] ipb: %d, num_instances: %d, q: %d, m: %d\n", ipb, num_instances, q, m);
    #endif

    // 4. one addition
    {
        // dry run
        if (v == 1)
            baddKer1<Base,m>      <<< grid, block >>>(d_as, d_bs, d_rs);
        else if (v == 2)
            baddKer2<Base,m,q>    <<< grid, block >>>(d_as, d_bs, d_rs);
        else // v == 3
            baddKer3<Base,m,q,ipb><<< grid, block >>>(d_as, d_bs, d_rs);
    
        hipDeviceSynchronize();
        gpuAssert( hipPeekAtLastError() );

        // timing instrumentation
        uint64_t elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 

        if (v == 1)
            for(int i=0; i<GPU_RUNS_ADD; i++)
                baddKer1<Base,m>      <<< grid, block >>>(d_as, d_bs, d_rs);
        else if (v == 2)
            for(int i=0; i<GPU_RUNS_ADD; i++)
                baddKer2<Base,m,q>    <<< grid, block >>>(d_as, d_bs, d_rs);
        else // v == 3
            for(int i=0; i<GPU_RUNS_ADD; i++)
                baddKer3<Base,m,q,ipb><<< grid, block >>>(d_as, d_bs, d_rs);

        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / GPU_RUNS_ADD;

        // print results
        gpuAssert( hipPeekAtLastError() );

        double runtime_microsecs = elapsed; 
        double bytes_accesses = 3.0 * num_instances * m * sizeof(uint_t);  
        double gigabytes = bytes_accesses / (runtime_microsecs * 1000);

        printf("One v%d addition  of %d-bit big integers (base u%d) runs %d instances \
in:\t%lu microsecs, GB/sec: %.2f, Mil-Instances/sec: %.2f\n", v, m*Base::bits, Base::bits,
               num_instances, elapsed, gigabytes, num_instances/runtime_microsecs);
        
        hipMemcpy(h_rs, d_rs, mem_size_nums, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }

    // 5. ten additions
    {
        // dry run
        if      (v == 1)
            baddKer1Bench<Base,m,10>      <<< grid, block >>>(d_as, d_bs, d_rs);
        else if (v == 2)
            baddKer2Bench<Base,m,q,10>    <<< grid, block >>>(d_as, d_bs, d_rs);
        else // (v == 3)
            baddKer3Bench<Base,m,q,ipb,10><<< grid, block >>>(d_as, d_bs, d_rs);
    
        hipDeviceSynchronize();
        gpuAssert( hipPeekAtLastError() );

        // timing instrumentation
        uint64_t elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 

        if      (v == 1)
            for(int i=0; i<GPU_RUNS_ADD; i++)
                baddKer1Bench<Base,m,10>      <<< grid, block >>>(d_as, d_bs, d_rs);
        else if (v == 2)
            for(int i=0; i<GPU_RUNS_ADD; i++)
                baddKer2Bench<Base,m,q,10>    <<< grid, block >>>(d_as, d_bs, d_rs);
        else // (v == 3)
            for(int i=0; i<GPU_RUNS_ADD; i++)
                baddKer3Bench<Base,m,q,ipb,10><<< grid, block >>>(d_as, d_bs, d_rs);

        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / GPU_RUNS_ADD;

        // print results
        gpuAssert( hipPeekAtLastError() );

        double runtime_microsecs = elapsed; 
        double bytes_accesses = 3.0 * num_instances * m * sizeof(uint_t);  
        double gigabytes = bytes_accesses / (runtime_microsecs * 1000);

        printf("Ten v%d additions of %d-bit big integers (base u%d) runs %d instances \
in:\t%lu microsecs, GB/sec: %.2f, Mil-Instances/sec: %.2f\n", v, m*Base::bits, Base::bits,
               num_instances, elapsed, gigabytes, num_instances/runtime_microsecs);
    }

    // 6. cleanup
    hipFree(d_as);
    hipFree(d_bs);
    hipFree(d_rs);
}

// performs big int addition using gmp
template<int m>
void gmpAdd(int num_instances, uint32_t* as, uint32_t* bs, uint32_t* rs) {
    uint32_t* it_as = as;
    uint32_t* it_bs = bs;
    uint32_t* it_rs = rs;
        
    for(int i=0; i<num_instances; i++) {
        gmpAddMulOnce<m>(true, it_as, it_bs, it_rs);
        it_as += m; it_bs += m; it_rs += m;
    }
}

// wrapper that benchmarks (and possible validates) the big int addition kernels;
// m is the size of the big word in u32 units.
template<class Base, int m>
void testAddition(int num_instances, uint64_t* h_as_64, uint64_t* h_bs_64,
                  uint64_t* h_rs_gmp_64, uint64_t* h_rs_our_64, uint32_t with_validation) {
    
    assert((Base::bits >= 32) && (Base::bits % 32 == 0));
    printf("\n");

    using uint_t = typename Base::uint_t;
    uint_t *h_as = (uint_t*) h_as_64;
    uint_t *h_bs = (uint_t*) h_bs_64;
    uint_t *h_rs_our = (uint_t*) h_rs_our_64;
    uint32_t *h_rs_gmp_32 = (uint32_t*) h_rs_gmp_64;

    if(with_validation)
        gmpAdd<m>(num_instances, (uint32_t*)h_as, (uint32_t*)h_bs, h_rs_gmp_32);

    gpuAdd<Base,m/(Base::bits/32),1>(num_instances, h_as, h_bs, h_rs_our);
    if(with_validation)
        validateExact(h_rs_gmp_32, (uint32_t*)h_rs_our, num_instances*m);

    gpuAdd<Base,m/(Base::bits/32),2>(num_instances, h_as, h_bs, h_rs_our);
    if(with_validation)
        validateExact(h_rs_gmp_32, (uint32_t*)h_rs_our, num_instances*m);

    gpuAdd<Base,m/(Base::bits/32),3>(num_instances, h_as, h_bs, h_rs_our);
    if(with_validation)
        validateExact(h_rs_gmp_32, (uint32_t*)h_rs_our, num_instances*m);
}

/*****************************************/
/*** Main program that runs test suits ***/
/*****************************************/

// runs the big int addition kernel benchmarks and tests.
template<typename Base>
void runAdditions(uint64_t total_work) {
    uint64_t *h_as, *h_bs, *h_rs_gmp, *h_rs_our;
    mkRandArrays<32,32>( total_work/32, &h_as, &h_bs, &h_rs_gmp, &h_rs_our );

    testAddition<Base, 2048>( total_work/2048, h_as, h_bs, h_rs_gmp, h_rs_our, WITH_VALIDATION );
    testAddition<Base, 1024>( total_work/1024, h_as, h_bs, h_rs_gmp, h_rs_our, WITH_VALIDATION );
    testAddition<Base, 512> ( total_work/512,  h_as, h_bs, h_rs_gmp, h_rs_our, WITH_VALIDATION );
    testAddition<Base, 256> ( total_work/256,  h_as, h_bs, h_rs_gmp, h_rs_our, WITH_VALIDATION );
    testAddition<Base, 128> ( total_work/128,  h_as, h_bs, h_rs_gmp, h_rs_our, WITH_VALIDATION );
    testAddition<Base, 64>  ( total_work/64,   h_as, h_bs, h_rs_gmp, h_rs_our, WITH_VALIDATION );
    testAddition<Base, 32>  ( total_work/32,   h_as, h_bs, h_rs_gmp, h_rs_our, WITH_VALIDATION );
    testAddition<Base, 16>  ( total_work/16,   h_as, h_bs, h_rs_gmp, h_rs_our, WITH_VALIDATION );

    free(h_as);
    free(h_bs);
    free(h_rs_gmp);
    free(h_rs_our);
}

int main(int argc, char * argv[]) {
    if (argc != 2) {
        printf("Usage: %s <batch-size>\n", argv[0]);
        exit(1);
    }
    const int total_work = atoi(argv[1]);

    runAdditions<U64bits>(total_work);
}
